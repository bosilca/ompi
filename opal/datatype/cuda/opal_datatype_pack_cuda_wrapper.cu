#include "opal/datatype/opal_convertor_internal.h"
#include "opal/datatype/opal_datatype_internal.h"
#include "opal/util/output.h"

#include "opal_datatype_cuda_internal.cuh"
#include "opal_datatype_cuda.cuh"

#include <stdio.h>
#include <assert.h>

int32_t opal_datatype_cuda_generic_simple_pack_function_iov( opal_convertor_t* pConvertor,
                                                             struct iovec* iov,
                                                             uint32_t* out_size,
                                                             size_t* max_data )
{
    size_t buffer_size;
    unsigned char *destination;
    size_t total_packed;
    uint8_t transfer_required, free_required;
    hipStream_t working_stream = NULL; 
    hipError_t cuda_err;

#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    TIMER_DATA_TYPE start, end, start_total, end_total;
    long total_time, move_time;
#endif

    if ((iov[0].iov_base == NULL) || opal_datatype_cuda_is_gpu_buffer(iov[0].iov_base)) {
        assert (iov[0].iov_len != 0);
        buffer_size = iov[0].iov_len;
        
        if (iov[0].iov_base == NULL) {
            iov[0].iov_base = (unsigned char *)opal_datatype_cuda_malloc_gpu_buffer(buffer_size, 0);
            destination = (unsigned char *)iov[0].iov_base;
            pConvertor->gpu_buffer_ptr = destination;
            pConvertor->gpu_buffer_size = buffer_size;
            free_required = 1;
        } else {
            destination = (unsigned char *)iov[0].iov_base;
            free_required = 0;
        }
        transfer_required = 0;
    } else {
        buffer_size = iov[0].iov_len;
        if (OPAL_DATATYPE_USE_ZEROCPY) {
            pConvertor->gpu_buffer_ptr = NULL;
            transfer_required = 0;
            free_required = 0;
            cuda_err = hipHostGetDevicePointer((void **)&destination, (void *)iov[0].iov_base, 0);
            if (cuda_err != hipSuccess) {
                OPAL_OUTPUT_VERBOSE((0, opal_datatype_cuda_output, "Zero copy is not supported\n"));
                return 0;
            }
        } else {
            if (pConvertor->gpu_buffer_ptr == NULL) {
                pConvertor->gpu_buffer_ptr = (unsigned char*)opal_datatype_cuda_malloc_gpu_buffer(buffer_size, 0);
                pConvertor->gpu_buffer_size = buffer_size;
            }
            transfer_required = 1;
            free_required = 1;
            destination = pConvertor->gpu_buffer_ptr + pConvertor->pipeline_size * pConvertor->pipeline_seq;
        }
    }   

    total_packed = 0;
    
#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start_total);
#endif
    
    /* start pack */
    if (cuda_iov_cache_enabled) {
        opal_datatype_cuda_generic_simple_pack_function_iov_cached(pConvertor, destination, buffer_size, &total_packed);
    } else {
        opal_datatype_cuda_generic_simple_pack_function_iov_non_cached(pConvertor, destination, buffer_size, &total_packed);
    }

    pConvertor->bConverted += total_packed;
    OPAL_OUTPUT_VERBOSE((2, opal_datatype_cuda_output, "Pack total packed %ld\n", total_packed));
    
#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif
    if (transfer_required) {
        if (pConvertor->stream == NULL) {
            ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;
            working_stream = cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id];
        } else {
            working_stream = (hipStream_t)pConvertor->stream;
        }
        cuda_err = hipMemcpyAsync(iov[0].iov_base, destination, total_packed, hipMemcpyDeviceToHost, working_stream);
        CUDA_ERROR_CHECK(cuda_err);
        if (!(pConvertor->flags & CONVERTOR_ACCELERATOR_ASYNC)) {
            cuda_err = hipStreamSynchronize(working_stream);
            CUDA_ERROR_CHECK(cuda_err);
        }
    } 
#if defined(OPAL_DATATYPE_CUDA_TIMING) 
    GET_TIME( end );
    move_time = ELAPSED_TIME( start, end );
    OPAL_OUTPUT_VERBOSE((2, opal_datatype_cuda_output, "[Timing]: DtoH memcpy in %ld microsec, transfer required %d, pipeline_size %lu, pipeline_seq %lu\n", move_time, transfer_required, pConvertor->pipeline_size, pConvertor->pipeline_seq ));
#endif

    iov[0].iov_len = total_packed;
    *max_data = total_packed;
    *out_size = 1;
    
#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    GET_TIME( end_total );
    total_time = ELAPSED_TIME( start_total, end_total );
    OPAL_OUTPUT_VERBOSE((2, opal_datatype_cuda_output, "[Timing]: total packing in %ld microsec, kernel %ld microsec\n", total_time, total_time-move_time ));
#endif
    
    if( pConvertor->bConverted == pConvertor->local_size ) {
        pConvertor->flags |= CONVERTOR_COMPLETED;
        if (pConvertor->gpu_buffer_ptr != NULL && free_required && !(pConvertor->flags & CONVERTOR_ACCELERATOR_ASYNC)) {
           opal_datatype_cuda_free_gpu_buffer(pConvertor->gpu_buffer_ptr, 0);
           pConvertor->gpu_buffer_ptr = NULL;
        }
        return 1;
    }        
    return 0; 
}

int32_t opal_datatype_cuda_generic_simple_pack_function_iov_non_cached( opal_convertor_t* pConvertor, unsigned char *destination, size_t buffer_size, size_t *total_packed)
{
    uint32_t nb_blocks, thread_per_block, nb_blocks_used;
    unsigned char *destination_base, *source_base;
    uint8_t buffer_isfull = 0;
    hipError_t cuda_err;
    ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;
    ddt_cuda_iov_dist_cached_t* cuda_iov_dist_h_current;
    ddt_cuda_iov_dist_cached_t* cuda_iov_dist_d_current;
    ddt_cuda_iov_pipeline_block_non_cached_t *cuda_iov_pipeline_block_non_cached;
    hipStream_t cuda_stream_iov = NULL;
    const struct iovec *ddt_iov = NULL;
    uint32_t ddt_iov_count = 0;
    size_t contig_disp = 0;
    uint32_t ddt_iov_start_pos, ddt_iov_end_pos, current_ddt_iov_pos;
    OPAL_PTRDIFF_TYPE ddt_extent;

#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    TIMER_DATA_TYPE start, end;
    long total_time;
#endif

    OPAL_OUTPUT_VERBOSE((2, opal_datatype_cuda_output, "Pack using IOV non cached, convertor %p, GPU base %p, pack to buffer %p\n", pConvertor, pConvertor->pBaseBuf, destination));
    
    opal_convertor_raw_cached( pConvertor, &ddt_iov, &ddt_iov_count);
    if (ddt_iov == NULL) {
        OPAL_OUTPUT_VERBOSE((0, opal_datatype_cuda_output, "Can not get ddt iov\n"));
        return OPAL_ERROR;
    }
    
    thread_per_block = CUDA_WARP_SIZE * 5;
    nb_blocks = 256;
    opal_datatype_type_extent(pConvertor->pDesc, &ddt_extent);
    source_base = (unsigned char*)pConvertor->pBaseBuf + pConvertor->current_count * ddt_extent; 
    destination_base = destination;
    
    while( pConvertor->current_count < pConvertor->count && !buffer_isfull) {
        
        nb_blocks_used = 0;
        ddt_iov_start_pos = pConvertor->current_iov_pos;
        ddt_iov_end_pos = ddt_iov_start_pos + IOV_PIPELINE_SIZE;
        if (ddt_iov_end_pos > ddt_iov_count) {
            ddt_iov_end_pos = ddt_iov_count;
        }
        cuda_iov_pipeline_block_non_cached = current_cuda_device->cuda_iov_pipeline_block_non_cached[current_cuda_device->cuda_iov_pipeline_block_non_cached_first_avail];
        if (pConvertor->stream == NULL) {
            cuda_iov_pipeline_block_non_cached->hip_stream = cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id];
        } else {
            cuda_iov_pipeline_block_non_cached->hip_stream = (hipStream_t)pConvertor->stream;
        }
        cuda_iov_dist_h_current = cuda_iov_pipeline_block_non_cached->cuda_iov_dist_non_cached_h;
        cuda_iov_dist_d_current = cuda_iov_pipeline_block_non_cached->cuda_iov_dist_non_cached_d;
        cuda_stream_iov = cuda_iov_pipeline_block_non_cached->hip_stream;
        cuda_err = hipEventSynchronize(cuda_iov_pipeline_block_non_cached->cuda_event);
        CUDA_ERROR_CHECK(cuda_err);

#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME(start);
#endif

        buffer_isfull = opal_datatype_cuda_iov_to_cuda_iov(pConvertor, ddt_iov, cuda_iov_dist_h_current, ddt_iov_start_pos, ddt_iov_end_pos, &buffer_size, &nb_blocks_used, total_packed, &contig_disp, &current_ddt_iov_pos);

#if defined(OPAL_DATATYPE_CUDA_TIMING)    
        GET_TIME( end );
        total_time = ELAPSED_TIME( start, end );
        OPAL_OUTPUT_VERBOSE((2, opal_datatype_cuda_output, "[Timing]: Pack src %p to dest %p, iov is prepared in %ld microsec, kernel submitted to CUDA stream %d, nb_blocks %d\n", source_base, destination_base, total_time,  cuda_streams->current_stream_id, nb_blocks_used));
#endif

        hipMemcpyAsync(cuda_iov_dist_d_current, cuda_iov_dist_h_current, sizeof(ddt_cuda_iov_dist_cached_t)*(nb_blocks_used+1), hipMemcpyHostToDevice, cuda_stream_iov);
        opal_generic_simple_pack_cuda_iov_cached_kernel<<<nb_blocks, thread_per_block, 0, cuda_stream_iov>>>(cuda_iov_dist_d_current, 0, nb_blocks_used, 0, 0, nb_blocks_used, source_base, destination_base);
        cuda_err = hipEventRecord(cuda_iov_pipeline_block_non_cached->cuda_event, cuda_stream_iov);
        CUDA_ERROR_CHECK(cuda_err);
        current_cuda_device->cuda_iov_pipeline_block_non_cached_first_avail ++;
        if (current_cuda_device->cuda_iov_pipeline_block_non_cached_first_avail >= NB_PIPELINE_NON_CACHED_BLOCKS) {
            current_cuda_device->cuda_iov_pipeline_block_non_cached_first_avail = 0;
        }
        destination_base += contig_disp;
        
        if (!buffer_isfull) {
            pConvertor->current_iov_pos = current_ddt_iov_pos;
            if (current_ddt_iov_pos == ddt_iov_count) {
                pConvertor->current_count ++;
                pConvertor->current_iov_pos = 0;
                source_base += ddt_extent;
            }
        }
        
    }
        
    return OPAL_SUCCESS;
}

int32_t opal_datatype_cuda_generic_simple_pack_function_iov_cached( opal_convertor_t* pConvertor, unsigned char *destination, size_t buffer_size, size_t *total_packed)
{
    uint32_t i;
    uint32_t nb_blocks, thread_per_block, nb_blocks_used;
    unsigned char *destination_base, *source_base;
    uint8_t buffer_isfull = 0;
    ddt_cuda_stream_t *cuda_streams = current_cuda_device->cuda_streams;
    hipStream_t cuda_stream_iov = NULL;
    uint32_t cuda_iov_start_pos, cuda_iov_end_pos;
    ddt_cuda_iov_total_cached_t* cached_cuda_iov = NULL;
    ddt_cuda_iov_dist_cached_t* cached_cuda_iov_dist_d = NULL;
    uint32_t *cached_cuda_iov_nb_bytes_list_h = NULL;
    uint32_t cached_cuda_iov_count = 0;
    opal_datatype_count_t convertor_current_count;
    OPAL_PTRDIFF_TYPE ddt_extent;

#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    TIMER_DATA_TYPE start, end;
    long total_time;
#endif
    
    OPAL_OUTPUT_VERBOSE((2, opal_datatype_cuda_output, "Pack using IOV cached, convertor %p, GPU base %p, pack to buffer %p\n", pConvertor, pConvertor->pBaseBuf, destination));

    destination_base = destination;
    thread_per_block = CUDA_WARP_SIZE * 8;
    nb_blocks = 64;
    source_base = (unsigned char*)pConvertor->pBaseBuf; 
    
    /* cuda iov is not cached, start to cache iov */
    if(opal_datatype_cuda_cuda_iov_is_cached(pConvertor) == 0) {
#if defined(OPAL_DATATYPE_CUDA_TIMING)
        GET_TIME(start);
#endif
        if (opal_datatype_cuda_cache_cuda_iov(pConvertor, &nb_blocks_used) == OPAL_SUCCESS) {
            OPAL_OUTPUT_VERBOSE((2, opal_datatype_cuda_output, "Pack cuda iov is cached, count %d\n", nb_blocks_used));
        } else {
            OPAL_OUTPUT_VERBOSE((0, opal_datatype_cuda_output, "Pack cache cuda iov is failed\n"));
            return OPAL_ERROR;
        }
#if defined(OPAL_DATATYPE_CUDA_TIMING)    
        GET_TIME( end );
        total_time = ELAPSED_TIME( start, end );
        OPAL_OUTPUT_VERBOSE((2, opal_datatype_cuda_output, "[Timing]: Pack cuda iov is cached in %ld microsec, nb_blocks %d\n", total_time, nb_blocks_used));
#endif
    }
    
    /* now we use cached cuda iov */
    opal_datatype_cuda_get_cached_cuda_iov(pConvertor, &cached_cuda_iov);
    cached_cuda_iov_dist_d = cached_cuda_iov->cuda_iov_dist_d;
    assert(cached_cuda_iov_dist_d != NULL);
    cached_cuda_iov_nb_bytes_list_h = cached_cuda_iov->nb_bytes_h;
    assert(cached_cuda_iov_nb_bytes_list_h != NULL);
    
    cached_cuda_iov_count = cached_cuda_iov->cuda_iov_count;
    cuda_iov_start_pos = pConvertor->current_cuda_iov_pos;
    cuda_iov_end_pos = cached_cuda_iov_count;
    nb_blocks_used = 0;
    if (pConvertor->stream == NULL) {
        cuda_stream_iov = cuda_streams->ddt_cuda_stream[cuda_streams->current_stream_id];
    } else {
        cuda_stream_iov = (hipStream_t)pConvertor->stream;
    }
    convertor_current_count = pConvertor->current_count;
   
#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif
    while( pConvertor->current_count < pConvertor->count && !buffer_isfull) {
        for (i = cuda_iov_start_pos; i < cuda_iov_end_pos && !buffer_isfull; i++) {
            if (buffer_size >= cached_cuda_iov_nb_bytes_list_h[i]) {
                *total_packed += cached_cuda_iov_nb_bytes_list_h[i];
                buffer_size -= cached_cuda_iov_nb_bytes_list_h[i];
                nb_blocks_used++;
            } else {
                buffer_isfull = 1;
                break;
            }
        }
        if (!buffer_isfull) {
            pConvertor->current_count ++;
            cuda_iov_start_pos = 0;
            cuda_iov_end_pos = cached_cuda_iov->cuda_iov_count;
        }
    }
#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    GET_TIME( end );
    total_time = ELAPSED_TIME( start, end );
    OPAL_OUTPUT_VERBOSE((2, opal_datatype_cuda_output, "[Timing]: Pack to dest %p, cached cuda iov is prepared in %ld microsec, kernel submitted to CUDA stream %d, nb_blocks %d\n", destination_base, total_time,  cuda_streams->current_stream_id, nb_blocks_used));
#endif
    opal_datatype_type_extent(pConvertor->pDesc, &ddt_extent);
    OPAL_OUTPUT_VERBOSE((2, opal_datatype_cuda_output, "Pack kernel launched src_base %p, dst_base %p, nb_blocks %d, extent %ld\n", source_base, destination_base, nb_blocks_used, ddt_extent));
#if defined(OPAL_DATATYPE_CUDA_TIMING)
    GET_TIME(start);
#endif    
    opal_generic_simple_pack_cuda_iov_cached_kernel<<<nb_blocks, thread_per_block, 0, cuda_stream_iov>>>(cached_cuda_iov_dist_d, pConvertor->current_cuda_iov_pos, cached_cuda_iov_count, ddt_extent, convertor_current_count, nb_blocks_used, source_base, destination_base);
    pConvertor->current_cuda_iov_pos += nb_blocks_used;
    pConvertor->current_cuda_iov_pos = pConvertor->current_cuda_iov_pos % cached_cuda_iov->cuda_iov_count;

#if defined(OPAL_DATATYPE_CUDA_TIMING)    
    GET_TIME( end );
    total_time = ELAPSED_TIME( start, end );
    OPAL_OUTPUT_VERBOSE((2, opal_datatype_cuda_output, "[Timing]: Pack kernel %ld microsec\n", total_time));
#endif    
    return OPAL_SUCCESS;
}

