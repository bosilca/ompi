#include "hip/hip_runtime.h"
#include "opal/datatype/opal_convertor_internal.h"
#include "opal/datatype/opal_datatype_internal.h"

#include "opal_datatype_cuda_internal.cuh"
#include <hip/hip_runtime.h>
#include <stdio.h> 

__global__ void opal_generic_simple_unpack_cuda_iov_cached_kernel( ddt_cuda_iov_dist_cached_t* cuda_iov_dist, uint32_t cuda_iov_pos, uint32_t cuda_iov_count, uint32_t ddt_extent, uint32_t current_count, int nb_blocks_used, unsigned char* destination_base, unsigned char* source_base, size_t cuda_iov_partial_length_start, size_t cuda_iov_partial_length_end)
{
    uint32_t i, j;
    size_t dst_offset, src_offset;
    unsigned char *_source_tmp, *_destination_tmp;
    uint32_t _nb_bytes;
    uint32_t current_cuda_iov_pos = cuda_iov_pos;
    size_t source_disp = cuda_iov_dist[current_cuda_iov_pos].contig_disp;
    size_t source_partial_disp = 0;
    size_t contig_disp; 
    uint32_t _my_cuda_iov_pos;
    uint32_t _my_cuda_iov_iteration;
    size_t ddt_size = cuda_iov_dist[cuda_iov_count].contig_disp;

    __shared__ uint32_t nb_tasks_per_block;
    __shared__ uint32_t WARP_SIZE;
    __shared__ uint32_t nb_warp_per_block;
    uint32_t copy_count;
    uint8_t alignment;
    uint64_t tmp_var_64[KERNEL_UNROLL];
    uint32_t tmp_var_32[KERNEL_UNROLL];
    unsigned char tmp_var_8[KERNEL_UNROLL];
    uint32_t u, k;
    uint32_t copy_count_16, copy_count_8, copy_count_left;
    
    if (threadIdx.x == 0) {
        nb_tasks_per_block = nb_blocks_used / gridDim.x;
        if (blockIdx.x < nb_blocks_used % gridDim.x) {
            nb_tasks_per_block ++;
        }
        WARP_SIZE = 32;
        nb_warp_per_block = blockDim.x / WARP_SIZE;
    }
    __syncthreads();
    
    const uint32_t warp_id_per_block = threadIdx.x / WARP_SIZE;
    const uint32_t tid_per_warp = threadIdx.x & (WARP_SIZE - 1);
    
    if (cuda_iov_partial_length_start != 0) {
        source_partial_disp = (cuda_iov_dist[current_cuda_iov_pos+1].contig_disp - cuda_iov_dist[current_cuda_iov_pos].contig_disp) - cuda_iov_partial_length_start;
    }
    
    for (i = warp_id_per_block; i < nb_tasks_per_block; i+= nb_warp_per_block) {
        /* these 3 variables are used multiple times, so put in in register */
        _my_cuda_iov_pos = (blockIdx.x + i * gridDim.x + current_cuda_iov_pos) % cuda_iov_count;
        _my_cuda_iov_iteration = (blockIdx.x + i * gridDim.x + current_cuda_iov_pos) / cuda_iov_count;
        contig_disp = cuda_iov_dist[_my_cuda_iov_pos].contig_disp; 
        
        src_offset = contig_disp + ddt_size * _my_cuda_iov_iteration - source_disp - source_partial_disp;
        dst_offset = cuda_iov_dist[_my_cuda_iov_pos].ncontig_disp + (_my_cuda_iov_iteration + current_count) * ddt_extent;
        _nb_bytes = cuda_iov_dist[_my_cuda_iov_pos + 1].contig_disp - contig_disp;

        if (i == 0 && blockIdx.x == 0 && cuda_iov_partial_length_start != 0) {
            src_offset = contig_disp + ddt_size * _my_cuda_iov_iteration - source_disp;
            dst_offset = dst_offset + _nb_bytes - cuda_iov_partial_length_start;  
            _nb_bytes = cuda_iov_partial_length_start;
        } else if (i == nb_tasks_per_block-1 && (blockIdx.x == (nb_blocks_used-1) % gridDim.x) && cuda_iov_partial_length_end != 0) {
            _nb_bytes = cuda_iov_partial_length_end;
        }
        
        _destination_tmp = destination_base + dst_offset; 
        _source_tmp = source_base + src_offset;
        if ((uintptr_t)(_destination_tmp) % ALIGNMENT_DOUBLE == 0 && (uintptr_t)(_source_tmp) % ALIGNMENT_DOUBLE == 0 && _nb_bytes % ALIGNMENT_DOUBLE == 0) {
            alignment = ALIGNMENT_DOUBLE;
        } else if ((uintptr_t)(_destination_tmp) % ALIGNMENT_FLOAT == 0 && (uintptr_t)(_source_tmp) % ALIGNMENT_FLOAT == 0 && _nb_bytes % ALIGNMENT_FLOAT == 0) {
            alignment = ALIGNMENT_FLOAT;
        } else {
            alignment = ALIGNMENT_CHAR;
        }
        copy_count = _nb_bytes / alignment;

        if (alignment == ALIGNMENT_DOUBLE) {
            uint64_t *_source_base_64, *_destination_base_64; 
            copy_count_16 = copy_count  / (WARP_SIZE * UNROLL_16) * (WARP_SIZE * UNROLL_16);
            _source_base_64 = (uint64_t *)(source_base + src_offset);
            _destination_base_64 = (uint64_t *)(destination_base + dst_offset);
            if (copy_count_16 > 0) {
                for (k = 0; k < copy_count_16; k += UNROLL_16 * WARP_SIZE) {
                    #pragma unroll
                    for (u = 0; u < UNROLL_16; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        tmp_var_64[u] = *(_source_base_64 + j);

                    }
                    #pragma unroll
                    for (u = 0; u < UNROLL_16; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        *(_destination_base_64 + j) = tmp_var_64[u];

                    }
                }
            }
            _source_base_64 += copy_count_16;
            _destination_base_64 += copy_count_16;
            
            copy_count_8 = (copy_count - copy_count_16) / (WARP_SIZE * UNROLL_8) * (WARP_SIZE * UNROLL_8);
            if (copy_count_8 > 0) {
                for (k = 0; k < copy_count_8; k += UNROLL_8 * WARP_SIZE) {
                    #pragma unroll
                    for (u = 0; u < UNROLL_8; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        tmp_var_64[u] = *(_source_base_64 + j);

                    }
                    #pragma unroll
                    for (u = 0; u < UNROLL_8; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        *(_destination_base_64 + j) = tmp_var_64[u];

                    }
                }
            }
            _source_base_64 += copy_count_8;
            _destination_base_64 += copy_count_8;
        
            copy_count_left = copy_count - copy_count_16 - copy_count_8;
            if (copy_count_left > 0) {
                #pragma unroll
                for (u = 0; u < UNROLL_8; u++) {
                    j = tid_per_warp + u * WARP_SIZE;
                    if (j < copy_count_left) {
                        tmp_var_64[u] = *(_source_base_64 + j);
                    } else {
                        break;
                    }
                }
                #pragma unroll
                for (u = 0; u < UNROLL_8; u++) {
                    j = tid_per_warp + u * WARP_SIZE;
                    if (j < copy_count_left) {
                        *(_destination_base_64 + j) = tmp_var_64[u];
                    } else {
                        break;
                    }
                }
            }
        } else if (alignment == ALIGNMENT_FLOAT) {
            uint32_t *_source_base_32, *_destination_base_32;    
            copy_count_16 = copy_count  / (WARP_SIZE * UNROLL_16) * (WARP_SIZE * UNROLL_16);
            _source_base_32 = (uint32_t *)(source_base + src_offset);
            _destination_base_32 = (uint32_t *)(destination_base + dst_offset);
            if (copy_count_16 > 0) {
                for (k = 0; k < copy_count_16; k += UNROLL_16 * WARP_SIZE) {
                    #pragma unroll
                    for (u = 0; u < UNROLL_16; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        tmp_var_32[u] = *(_source_base_32 + j);

                    }
                    #pragma unroll
                    for (u = 0; u < UNROLL_16; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        *(_destination_base_32 + j) = tmp_var_32[u];

                    }
                }
            }
            _source_base_32 += copy_count_16;
            _destination_base_32 += copy_count_16;
        
            copy_count_8 = (copy_count - copy_count_16) / (WARP_SIZE * UNROLL_8) * (WARP_SIZE * UNROLL_8);
            if (copy_count_8 > 0) {
                for (k = 0; k < copy_count_8; k += UNROLL_8 * WARP_SIZE) {
                    #pragma unroll
                    for (u = 0; u < UNROLL_8; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        tmp_var_32[u] = *(_source_base_32 + j);

                    }
                    #pragma unroll
                    for (u = 0; u < UNROLL_8; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        *(_destination_base_32 + j) = tmp_var_32[u];

                    }
                }
            }
            _source_base_32 += copy_count_8;
            _destination_base_32 += copy_count_8;
        
            copy_count_left = copy_count - copy_count_16 - copy_count_8;
            if (copy_count_left > 0) {
                #pragma unroll
                for (u = 0; u < UNROLL_8; u++) {
                    j = tid_per_warp + u * WARP_SIZE;
                    if (j < copy_count_left) {
                        tmp_var_32[u] = *(_source_base_32 + j);
                    } else {
                        break;
                    }
                }
                #pragma unroll
                for (u = 0; u < UNROLL_8; u++) {
                    j = tid_per_warp + u * WARP_SIZE;
                    if (j < copy_count_left) {
                        *(_destination_base_32 + j) = tmp_var_32[u];
                    } else {
                        break;
                    }
                }
            }
        } else {
            unsigned char *_source_base_8, *_destination_base_8;
        
            copy_count_16 = copy_count  / (WARP_SIZE * UNROLL_16) * (WARP_SIZE * UNROLL_16);
            _source_base_8 = (unsigned char *)(source_base + src_offset);
            _destination_base_8 = (unsigned char *)(destination_base + dst_offset);
            if (copy_count_16 > 0) {
                for (k = 0; k < copy_count_16; k += UNROLL_16 * WARP_SIZE) {
                    #pragma unroll
                    for (u = 0; u < UNROLL_16; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        tmp_var_8[u] = *(_source_base_8 + j);

                    }
                    #pragma unroll
                    for (u = 0; u < UNROLL_16; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        *(_destination_base_8 + j) = tmp_var_8[u];

                    }
                }
            }
            _source_base_8 += copy_count_16;
            _destination_base_8 += copy_count_16;
        
            copy_count_8 = (copy_count - copy_count_16) / (WARP_SIZE * UNROLL_8) * (WARP_SIZE * UNROLL_8);
            if (copy_count_8 > 0) {
                for (k = 0; k < copy_count_8; k += UNROLL_8 * WARP_SIZE) {
                    #pragma unroll
                    for (u = 0; u < UNROLL_8; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        tmp_var_8[u] = *(_source_base_8 + j);

                    }
                    #pragma unroll
                    for (u = 0; u < UNROLL_8; u++) {
                        j = tid_per_warp + u * WARP_SIZE + k;
                        *(_destination_base_8 + j) = tmp_var_8[u];

                    }
                }
            }
            _source_base_8 += copy_count_8;
            _destination_base_8 += copy_count_8;
        
            copy_count_left = copy_count - copy_count_16 - copy_count_8;
            if (copy_count_left > 0) {
                #pragma unroll
                for (u = 0; u < UNROLL_8; u++) {
                    j = tid_per_warp + u * WARP_SIZE;
                    if (j < copy_count_left) {
                        tmp_var_8[u] = *(_source_base_8 + j);
                    } else {
                        break;
                    }
                }
                #pragma unroll
                for (u = 0; u < UNROLL_8; u++) {
                    j = tid_per_warp + u * WARP_SIZE;
                    if (j < copy_count_left) {
                        *(_destination_base_8 + j) = tmp_var_8[u];
                    } else {
                        break;
                    }
                }
            }
        }
    }
}
